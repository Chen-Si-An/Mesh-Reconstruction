#include "hip/hip_runtime.h"
/**
 * Author: rodrigo
 * 2015
 */
#include "GpuRoutines.h"

#include <ostream>
#include <iostream>
#include "GpuUtils.h"

#define EPSILON		1E-10
#define MAX_BLOCKS	65535
#define MAX_THREADS	1024

// Pointers to memory in device
bool *devNotUsed = NULL;
gpu::Point *devPoints = NULL;
gpu::DeviceNode *devNodes = NULL;
gpu::BallCenter *devAuxCenter = NULL;
gpu::DeviceKDTree *devAuxTree = NULL;

// Global variable in device
__device__ gpu::BallCenter *devCenter;
__device__ gpu::DeviceKDTree *devKDTree;

// Debug variables
__device__ int devFound;
__device__ int devTreeSize;
__device__ int devTreeRoot;
__device__ int devLeftChild;
__device__ int devRightChild;

std::ostream &operator<<(std::ostream &_stream, const gpu::BallCenter &_center)
{
	_stream << "c=(" << _center.cx << ", " << _center.cy << ", " << _center.cz << ") / (" << _center.idx0 << ", " << _center.idx1 << ", " << _center.idx2 << ")";
	return _stream;
}

//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
/*
void GpuRoutines::allocPoints(const pcl::PointCloud<pcl::PointNormal>::Ptr &_cloud)
{
	gpu::Point *hostPoints = (gpu::Point *) &_cloud->points[0];
	GpuUtils::createInDev<gpu::Point>(&devPoints, hostPoints, _cloud->size());
}

void GpuRoutines::allocUsed(const pcl::PointCloud<pcl::PointNormal>::Ptr &_cloud, const bool* _notUsed)
{
	GpuUtils::createInDev<bool>(&devNotUsed, _notUsed, _cloud->size());
}
*/
void GpuRoutines::allocPoints(const std::vector<gpu::Point> *_cloud)
{
	gpu::Point *hostPoints = (gpu::Point *) &_cloud->at(0);
	GpuUtils::createInDev<gpu::Point>(&devPoints, hostPoints, (int)_cloud->size());
}

void GpuRoutines::allocUsed(const std::vector<gpu::Point> *_cloud, const bool* _notUsed)
{
	GpuUtils::createInDev<bool>(&devNotUsed, _notUsed, (int)_cloud->size());
}
//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t

__device__ bool isOriented(const gpu::Point *_normal, const gpu::Point *_p0, const gpu::Point *_p1, const gpu::Point *_p2)
{
	int count = 0;
	count = _p0->normalDot(*_normal) < 0 ? count + 1 : count;
	count = _p1->normalDot(*_normal) < 0 ? count + 1 : count;
	count = _p2->normalDot(*_normal) < 0 ? count + 1 : count;

	return count <= 1;
}

__device__ float getCircumscribedCircle(const gpu::Point &_p0, const gpu::Point &_p1, const gpu::Point &_p2, gpu::BallCenter &_center)
{
	gpu::Point d10 = _p1 - _p0;
	gpu::Point d20 = _p2 - _p0;
	gpu::Point d01 = _p0 - _p1;
	gpu::Point d12 = _p1 - _p2;
	gpu::Point d21 = _p2 - _p1;
	gpu::Point d02 = _p0 - _p2;

	float norm01 = d01.norm();
	float norm12 = d12.norm();
	float norm02 = d02.norm();

	float norm01C12 = d01.cross(d12).norm();

	float alpha = (norm12 * norm12 * d01.dot(d02)) / (2 * norm01C12 * norm01C12);
	float beta = (norm02 * norm02 * d10.dot(d12)) / (2 * norm01C12 * norm01C12);
	float gamma = (norm01 * norm01 * d20.dot(d21)) / (2 * norm01C12 * norm01C12);

	gpu::Point circumscribedCircleCenter = (_p0 * alpha) + (_p1 * beta) + (_p2 * gamma);
	float circumscribedCircleRadius = (norm01 * norm12 * norm02) / (2 * norm01C12);

	_center.cx = circumscribedCircleCenter.x;
	_center.cy = circumscribedCircleCenter.y;
	_center.cz = circumscribedCircleCenter.z;

	return circumscribedCircleRadius;
}

__device__ bool getBallCenter(const gpu::Point *_point0, const gpu::Point *_point1, const gpu::Point *_point2, const float _ballRadius, gpu::BallCenter *_center)
{
	bool status = false;
	_center->isValid = false;

	// Local pointers
	const gpu::Point *p0 = _point0;
	const gpu::Point *p1 = _point1;
	const gpu::Point *p2 = _point2;

	gpu::Point v10 = p1->operator -(*p0);
	gpu::Point v20 = p2->operator -(*p0);
	gpu::Point normal = v10.cross(v20);

	// Calculate ball center only if points are not collinear
	if (normal.norm() > EPSILON)
	{
		// Normalize to avoid precision errors while checking the orientation
		normal.normalize();
		if (!isOriented(&normal, p0, p1, p2))
		{
			// Wrong orientation, swap vertices to get a CCW oriented triangle so face's normal pointing upwards
			int aux = _center->idx0;
			_center->idx0 = _center->idx1;
			_center->idx1 = aux;

			p0 = _point1;
			p1 = _point0;

			v10 = p1->operator -(*p0);
			v20 = p2->operator -(*p0);
			normal = v10.cross(v20);
			normal.normalize();
		}

		float circleRadius = getCircumscribedCircle(*p0, *p1, *p2, *_center);
		float squaredDistance = _ballRadius * _ballRadius - circleRadius * circleRadius;

		if (squaredDistance > 0)
		{
			float distance = sqrt(fabs(squaredDistance));
			_center->add(normal * distance);
			_center->isValid = true;
			status = true;
		}
	}

	return status;
}

__device__ bool isEmpty(const gpu::BallCenter *_center, const gpu::Point *_points, const int _pointNumber, const float _ballRadius)
{
	for (int i = 0; i < _pointNumber; i++)
	{
		if ((i == _center->idx0) || (i == _center->idx1) || (i == _center->idx2))
			continue;

		if (_center->dist(_points[i]) >= _ballRadius)
			continue;

		return false;
	}

	return true;
}

__global__ void checkForSeeds(const gpu::Point *_points, const int _pointNumber, const int *_neighbors, const int _neighborsSize, const bool *_notUsed, const int _index0, const float _ballRadius, const int _neighborsPerThread)
{
	///// Assign debug variables /////
	/*devTreeSize = devKDTree->size;
	 devTreeRoot = devKDTree->root;
	 devLeftChild = devKDTree->nodes[devKDTree->root].left;
	 devRightChild = devKDTree->nodes[devKDTree->root].right;*/
	//////////////////////////////////
	int start0 = blockIdx.x;
	int end0 = start0 + 1;

	int start1 = threadIdx.x * _neighborsPerThread;
	int end1 = start1 + _neighborsPerThread;

	for (int j = start0; j < end0 && j < _neighborsSize; j++)
	{
		if (devFound == 0)
		{
			int index1 = _neighbors[j];

			// Skip invalid combinations
			if (index1 == _index0 || !_notUsed[index1])
				continue;

			for (size_t k = start1; k < end1 && k < _neighborsSize && devFound == 0; k++)
			{
				int index2 = _neighbors[k];

				// Skip invalid combinations
				if (index1 == index2 || index2 == _index0 || !_notUsed[index2])
					continue;

				gpu::BallCenter center(_index0, index1, index2);
				if (getBallCenter(&_points[_index0], &_points[index1], &_points[index2], _ballRadius, &center))
				{
					//if (isEmpty(&center, _points, _pointNumber, _ballRadius))
					if (devKDTree->isEmptyRadius(&center, _ballRadius))
					{
						if (devFound == 0)
						{
							atomicExch(&devFound, 1);
							devCenter->cx = center.cx;
							devCenter->cy = center.cy;
							devCenter->cz = center.cz;
							devCenter->idx0 = center.idx0;
							devCenter->idx1 = center.idx1;
							devCenter->idx2 = center.idx2;
							devCenter->isValid = center.isValid;
							break;
						}
					}
				}
			}
		}
	}
}

//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
//gpu::BallCenter GpuRoutines::findSeed(const pcl::PointCloud<pcl::PointNormal>::Ptr &_cloud, const std::vector<int> &_neighbors, const bool *_notUsed, const int _index0, const float _ballRadius)
gpu::BallCenter GpuRoutines::findSeed(const std::vector<gpu::Point> *_cloud, const std::vector<int> &_neighbors, const bool *_notUsed, const int _index0, const float _ballRadius)
//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
{
	size_t neighborsSize = _neighbors.size();
	if (neighborsSize > MAX_BLOCKS)
	{
		//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
		/*std::cout << "ERROR: radius size too big" << std::endl;
		exit(1);*/
		return gpu::BallCenter();
		//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
	}

	int blocks = neighborsSize < MAX_BLOCKS ? (int)neighborsSize : MAX_BLOCKS;
	int threads = neighborsSize < MAX_THREADS ? (int)neighborsSize : MAX_THREADS;
	int neighborsPerThread = (int)ceil((double) neighborsSize / threads);

	size_t cloudSize = _cloud->size();

	// Prepare memory buffers
	if (devPoints == NULL)
		allocPoints(_cloud);
	if (devNotUsed == NULL)
		allocUsed(_cloud, _notUsed);

	// Copy not used data to dev
	size_t notUsedBytes = sizeof(bool) * _cloud->size();
	GpuUtils::setData<bool>(&devNotUsed, _notUsed, (int)_cloud->size());

	// Create and prepare buffer with neighbors indices
	int *devNeighbors;
	GpuUtils::createInDev<int>(&devNeighbors, &_neighbors[0], (int)neighborsSize);

	// Prepare global variable 'devFound'
	int found = 0;
	GpuUtils::setSymbol<int>(devFound, &found);

	// Prepare global variable 'devFoundCenter'
	gpu::BallCenter center = gpu::BallCenter();
	if (devAuxCenter == NULL)
		GpuUtils::allocMemory<gpu::BallCenter>(&devAuxCenter, 1);
	GpuUtils::setData<gpu::BallCenter>(&devAuxCenter, &center, 1);
	GpuUtils::setSymbol<gpu::BallCenter *>(devCenter, &devAuxCenter);

	// Execute kernel
	checkForSeeds<<<blocks, threads>>>(devPoints, (int)_cloud->size(), devNeighbors, (int)neighborsSize, devNotUsed, _index0, _ballRadius, neighborsPerThread);

	///// Retrieve debug variables /////
//	hipMemcpyFromSymbol(&found, HIP_SYMBOL(devFound), sizeof(int));
//	checkErrors("hipMemcpyFromSymbol failed");
//
//	 int treeSize = -1;
//	 hipMemcpyFromSymbol(&treeSize, HIP_SYMBOL(devTreeSize), sizeof(int));
//	 checkErrors("hipMemcpyFromSymbol failed");
//
//	 int treeRoot = -5;
//	 hipMemcpyFromSymbol(&treeRoot, HIP_SYMBOL(devTreeRoot), sizeof(int));
//	 checkErrors("hipMemcpyFromSymbol failed");
//
//	 int leftChild = -5;
//	 hipMemcpyFromSymbol(&leftChild, HIP_SYMBOL(devLeftChild), sizeof(int));
//	 checkErrors("hipMemcpyFromSymbol failed");
//
//	 int rightChild = -5;
//	 hipMemcpyFromSymbol(&rightChild, HIP_SYMBOL(devRightChild), sizeof(int));
//	 checkErrors("hipMemcpyFromSymbol failed");
	////////////////////////////////////

	// Retrieve results
	GpuUtils::getData<gpu::BallCenter>(&center, devAuxCenter, 1);

	// Free allocated memory
	hipFree(devNeighbors);
	//checkErrors("hipFree devNeighbors failed");	//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t

	return center;
}

//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
//void GpuRoutines::buildInDeviceKDTree(const pcl::PointCloud<pcl::PointNormal>::Ptr &_cloud)
void GpuRoutines::buildInDeviceKDTree(const std::vector<gpu::Point> *_cloud)
//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
{
	// Build a KDTree using host memory
	//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
	/*
	gpu::HostKDTree tree = gpu::HostKDTree((const gpu::Point *) &_cloud->points[0]);
	for (size_t i = 0; i < _cloud->size(); i++)
		tree.insert((gpu::Point *) &_cloud->points[i], i);
	*/
	gpu::HostKDTree tree = gpu::HostKDTree((const gpu::Point *) &_cloud->at(0));
	for (size_t i = 0; i < _cloud->size(); i++)
		tree.insert((gpu::Point *) &_cloud->at(i), (int)i);
	//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t

	//*
	// Get the serialized version of the tree
	gpu::DeviceNode *hostMem = new gpu::DeviceNode[tree.size];
	tree.getSerializedRepresentation(hostMem);

	// Allocate cloud data if it has not already been done
	if (devPoints == NULL)
		allocPoints(_cloud);

	// Allocate memory for tree's nodes and copy data
	GpuUtils::createInDev<gpu::DeviceNode>(&devNodes, hostMem, tree.size);
	delete hostMem;	//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t

	// Create the serialized tree
	gpu::DeviceKDTree serializedTree = gpu::DeviceKDTree();
	serializedTree.root = 0;
	serializedTree.size = tree.size;
	serializedTree.nodes = devNodes;
	serializedTree.points = devPoints;

	// Allocate memory for the tree itself
	GpuUtils::createInDev<gpu::DeviceKDTree>(&devAuxTree, &serializedTree, 1);
	GpuUtils::setSymbol<gpu::DeviceKDTree *>(devKDTree, &devAuxTree);
	//*/
}

void GpuRoutines::releaseMemory()
{
	hipFree(devNotUsed);
	hipFree(devPoints);
	hipFree(devNodes);
	hipFree(devAuxCenter);
	hipFree(devAuxTree);

	//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
	devNotUsed = NULL;
	devPoints = NULL;
	devNodes = NULL;
	devAuxCenter = NULL;
	devAuxTree = NULL;
	//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
}

void GpuRoutines::prepareStackSize()
{
	size_t sizeLimit = 1024 * 5;
	hipDeviceSetLimit(hipLimitStackSize, sizeLimit);
	//checkErrors("hipDeviceSetLimit failed");	//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
}

//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
__global__ void searchCloserPoints(const gpu::Point _target, const gpu::Point *_points, const int _pointNumber, const double _searchRadius, const int _pointsPerThread, bool *_selected)
{
	int startIdx = (blockIdx.x * blockDim.x + threadIdx.x) * _pointsPerThread;
	double sqrRadius = _searchRadius * _searchRadius;


	for (int i = startIdx; i < startIdx + _pointsPerThread && i < _pointNumber; i++)
	{
		_selected[i] = _target.sqrDist(_points[i]) < sqrRadius;
	}
}

bool GpuRoutines::radiusSearch(const gpu::Point &_target, const std::vector<gpu::Point> *_cloud, double _radius, std::vector<int> &_idxs)
{
	/*int blocks = 10;
	int threads = 256;*/
	size_t cloudSize = _cloud->size();
	int blocks = cloudSize < MAX_BLOCKS ? (int)cloudSize : MAX_BLOCKS;
	int threads = cloudSize < MAX_THREADS ? (int)cloudSize : MAX_THREADS;

	// Copy points to device
	if (devPoints == NULL)
		allocPoints(_cloud);

	// Array to store points within radius
	bool *devSelected = NULL;
	GpuUtils::allocMemory<bool>(&devSelected, (int)cloudSize);
	//cudaCheckErrors("hipMalloc selected failed");

	// Calculate adequate number of blocks and threads
	while (cloudSize / blocks < 2)
		blocks /= 2;
	int pointsPerBlock = (int)ceil((double)cloudSize / blocks);

	while (pointsPerBlock / threads < 1)
		threads /= 2;
	int pointsPerThread = (int)ceil((double)pointsPerBlock / threads);

	// Execute kernel
	searchCloserPoints<<<blocks, threads>>>(_target, devPoints, (int)cloudSize, _radius, pointsPerThread, devSelected);

	// Copy data to host
	bool *selected = new bool[cloudSize];
	GpuUtils::getData<bool>(selected, devSelected, (int)cloudSize);
	//cudaCheckErrors("hipMemcpy selected failed");
	hipFree(devSelected);
	//cudaCheckErrors("hipFree selected failed");

	for (size_t i = 0; i < cloudSize; i++)
		if (selected[i])
			_idxs.push_back((int)i);

	delete(selected);

	return true;
}
//Allen_20221006A_Ball Pivoting�ϥ�GPU�[�t
